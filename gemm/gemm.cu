// optimize sgemm

#include <stdio.h>
#include <stdlib.h>
#include "assert.h" 

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>

// cal offset from row col and ld , in row-major matrix, ld is the width of the matrix
#define OFFSET(row, col, ld) ((row) * (ld) + (col))

// transfer float4
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

#define checkCudaErrors(func)				\
{									\
    hipError_t e = (func);			\
    if(e != hipSuccess)						                \
        printf ("%s %d CUDA: %s\n", __FILE__,  __LINE__, hipGetErrorString(e));		\
}

#define BLOCK_SIZE 32

__global__ void gemm_naive(float *A, float *B, float *C,
                      const int M,const int N, const int K) {
  int _m = blockIdx.x * BLOCK_SIZE + threadIdx.x;
  int _n = blockIdx.y * BLOCK_SIZE + threadIdx.y;
  if (_m < M and _n < N) {
    float sum = 0.f;
    for (int i = 0; i < K; ++i) {
      sum += A[_m * K + i] * B[i * N + _n];
    }
    C[_m * N + _n] = sum;
  }
}

__global__ void gemm_base(float *A, float *B, float *C,
                          const int M,const int N, const int K) {
  // Block index
  int bx = blockIdx.x;
  int by = blockIdx.y;

  // Thread index
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  unsigned int wA = K;
  unsigned int wB = N;

  // Index of the first sub-matrix of A processed by the block
  int aBegin = wA * BLOCK_SIZE * by;

  // Index of the last sub-matrix of A processed by the block
  int aEnd = aBegin + wA - 1;

  // Step size used to iterate through the sub-matrices of A
  int aStep = BLOCK_SIZE;

  // Index of the first sub-matrix of B processed by the block
  int bBegin = BLOCK_SIZE * bx;

  // Step size used to iterate through the sub-matrices of B
  int bStep = BLOCK_SIZE * wB;

  // Csub is used to store the element of the block sub-matrix
  // that is computed by the thread
  float Csub = 0;

  // Loop over all the sub-matrices of A and B
  // required to compute the block sub-matrix
  for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
    // Declaration of the shared memory array As used to
    // store the sub-matrix of A
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

    // Declaration of the shared memory array Bs used to
    // store the sub-matrix of B
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Load the matrices from device memory
    // to shared memory; each thread loads
    // one element of each matrix
    As[ty][tx] = A[a + wA * ty + tx];
    Bs[ty][tx] = B[b + wB * ty + tx];

    // Synchronize to make sure the matrices are loaded
    __syncthreads();

    // Multiply the two matrices together;
    // each thread computes one element
    // of the block sub-matrix
#pragma unroll

    for (int k = 0; k < BLOCK_SIZE; ++k) {
      Csub += As[ty][k] * Bs[k][tx];
    }

    // Synchronize to make sure that the preceding
    // computation is done before loading two new
    // sub-matrices of A and B in the next iteration
    __syncthreads();
  }

  // Write the block sub-matrix to device memory;
  // each thread writes one element
  int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
  C[c + wB * ty + tx] = Csub;
}

// K: ldA
// N: ldB
template <
    const int BLOCK_SIZE_M,  // height of block of C that each thread block calculate
    const int BLOCK_SIZE_K,  // width of block of A that each thread block load into shared memory
    const int BLOCK_SIZE_N,  // width of block of C that each thread block calculate
    const int THREAD_SIZE_Y, // height of block of C that each thread calculate
    const int THREAD_SIZE_X,  // width of block of C that each thread calculate
    const bool ENABLE_DOUBLE_BUFFER // whether enable double buffering or not
    > 
__global__ void gemm_v1( 
    float * __restrict__ A,
    float * __restrict__ B,
    float * __restrict__ C, 
    const int M,
    const int N,
    const int K) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    // the threads number in Block of X,Y
    const int THREAD_X_PER_BLOCK = BLOCK_SIZE_N / THREAD_SIZE_X;
    const int THREAD_Y_PER_BLOCK = BLOCK_SIZE_M / THREAD_SIZE_Y;
    const int THREAD_NUM_PER_BLOCK = THREAD_X_PER_BLOCK * THREAD_Y_PER_BLOCK;

    // thread id in cur Block
    const int tid = ty * THREAD_X_PER_BLOCK + tx;

    // shared memory
    __shared__ float As[2][BLOCK_SIZE_K][BLOCK_SIZE_M];
    __shared__ float Bs[2][BLOCK_SIZE_K][BLOCK_SIZE_N];
    // registers for C
    float accum[THREAD_SIZE_Y][THREAD_SIZE_X] = {0};
    // registers for A and B
    float frag_a[2][THREAD_SIZE_Y];
    float frag_b[2][THREAD_SIZE_X];
    // registers load global memory
    const int ldg_num_a = BLOCK_SIZE_M * BLOCK_SIZE_K / (THREAD_NUM_PER_BLOCK * 4);
    const int ldg_num_b = BLOCK_SIZE_K * BLOCK_SIZE_N / (THREAD_NUM_PER_BLOCK * 4);
    float ldg_a_reg[4*ldg_num_a];
    float ldg_b_reg[4*ldg_num_b];

    // threads number in one row
    const int A_TILE_THREAD_PER_ROW = BLOCK_SIZE_K / 4;
    const int B_TILE_THREAD_PER_ROW = BLOCK_SIZE_N / 4;

    // row number and col number that needs to be loaded by this thread
    const int A_TILE_ROW_START = tid / A_TILE_THREAD_PER_ROW;
    const int B_TILE_ROW_START = tid / B_TILE_THREAD_PER_ROW;

    const int A_TILE_COL = tid % A_TILE_THREAD_PER_ROW * 4; 
    const int B_TILE_COL = tid % B_TILE_THREAD_PER_ROW * 4;

    // row stride that thread uses to load multiple rows of a tile
    const int A_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / A_TILE_THREAD_PER_ROW;
    const int B_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / B_TILE_THREAD_PER_ROW;

    A = &A[(BLOCK_SIZE_M * by)* K];
    B = &B[BLOCK_SIZE_N * bx];

    //transfer first tile from global mem to shared mem
    // load A from global memory to shared memory
    #pragma unroll
    for ( int i = 0 ; i < BLOCK_SIZE_M ; i += A_TILE_ROW_STRIDE) {
        int ldg_index = i / A_TILE_ROW_STRIDE * 4;
        FETCH_FLOAT4(ldg_a_reg[ldg_index]) = FETCH_FLOAT4(A[OFFSET(
            A_TILE_ROW_START + i, // row
            A_TILE_COL, // col
            K )]);
        As[0][A_TILE_COL][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index];
        As[0][A_TILE_COL+1][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+1];
        As[0][A_TILE_COL+2][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+2];
        As[0][A_TILE_COL+3][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+3];
    }
    // load B from global memory to shared memory
    #pragma unroll
    for ( int i = 0 ; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
        FETCH_FLOAT4(Bs[0][B_TILE_ROW_START + i][B_TILE_COL]) = FETCH_FLOAT4(B[OFFSET(
                B_TILE_ROW_START + i, // row
                B_TILE_COL, // col
                N )]);
    }
    __syncthreads();
    // load A from shared memory to register
    #pragma unroll
    for (int thread_y = 0; thread_y < THREAD_SIZE_Y; thread_y += 4) {
        FETCH_FLOAT4(frag_a[0][thread_y]) = FETCH_FLOAT4(As[0][0][THREAD_SIZE_Y * ty + thread_y]);
    }
    // load B from shared memory to register
    #pragma unroll
    for (int thread_x = 0; thread_x < THREAD_SIZE_X; thread_x += 4) {
        FETCH_FLOAT4(frag_b[0][thread_x]) = FETCH_FLOAT4(Bs[0][0][THREAD_SIZE_X * tx + thread_x]);
    }

    int write_stage_idx = 1;
    int tile_idx = 0;
    do{
        tile_idx += BLOCK_SIZE_K;
        // load next tile from global mem
        if(tile_idx< K){
            #pragma unroll
            for ( int i = 0 ; i < BLOCK_SIZE_M ; i += A_TILE_ROW_STRIDE) {
                int ldg_index = i / A_TILE_ROW_STRIDE * 4;
                FETCH_FLOAT4(ldg_a_reg[ldg_index]) = FETCH_FLOAT4(A[OFFSET(
                    A_TILE_ROW_START + i, // row
                    A_TILE_COL + tile_idx, // col
                    K )]);
            }
            #pragma unroll
            for ( int i = 0 ; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
                int ldg_index = i / B_TILE_ROW_STRIDE * 4;
                FETCH_FLOAT4(ldg_b_reg[ldg_index]) = FETCH_FLOAT4(B[OFFSET(
                    tile_idx + B_TILE_ROW_START + i, // row
                    B_TILE_COL, // col
                    N )]);
            }
        }

        int load_stage_idx = write_stage_idx ^ 1;

        #pragma unroll
        for(int j=0; j<BLOCK_SIZE_K-1; ++j){
            // load next tile from shared mem to register 
            // load A from shared memory to register
            #pragma unroll
            for (int thread_y = 0; thread_y < THREAD_SIZE_Y; thread_y += 4) {
                FETCH_FLOAT4(frag_a[(j+1)%2][thread_y]) = FETCH_FLOAT4(As[load_stage_idx][j+1][THREAD_SIZE_Y * ty + thread_y]);
            }
            // load B from shared memory to register
            #pragma unroll
            for (int thread_x = 0; thread_x < THREAD_SIZE_X; thread_x += 4) {
                FETCH_FLOAT4(frag_b[(j+1)%2][thread_x]) = FETCH_FLOAT4(Bs[load_stage_idx][j+1][THREAD_SIZE_X * tx + thread_x]);
            }
            // compute C THREAD_SIZE_X x THREAD_SIZE_Y
            #pragma unroll
            for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) {
                #pragma unroll
                for (int thread_x = 0; thread_x < THREAD_SIZE_X; ++thread_x) {
                    accum[thread_y][thread_x] += frag_a[j%2][thread_y] * frag_b[j%2][thread_x];
                }
            }
        }

        if(tile_idx < K){
            #pragma unroll
            for ( int i = 0 ; i < BLOCK_SIZE_M ; i += A_TILE_ROW_STRIDE) {
                int ldg_index = i / A_TILE_ROW_STRIDE * 4;
                As[write_stage_idx][A_TILE_COL][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index];
                As[write_stage_idx][A_TILE_COL+1][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+1];
                As[write_stage_idx][A_TILE_COL+2][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+2];
                As[write_stage_idx][A_TILE_COL+3][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+3];
            }
            // load B from global memory to shared memory
            #pragma unroll
            for ( int i = 0 ; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
                int ldg_index = i / B_TILE_ROW_STRIDE * 4;
                FETCH_FLOAT4(Bs[write_stage_idx][B_TILE_ROW_START + i][B_TILE_COL]) = FETCH_FLOAT4(ldg_b_reg[ldg_index]);
            }
            // use double buffer, only need one sync
            __syncthreads();
            // switch
            write_stage_idx ^= 1;
        }

        // load first tile from shared mem to register of next iter
        // load A from shared memory to register
        #pragma unroll
        for (int thread_y = 0; thread_y < THREAD_SIZE_Y; thread_y += 4) {
            FETCH_FLOAT4(frag_a[0][thread_y]) = FETCH_FLOAT4(As[load_stage_idx^1][0][THREAD_SIZE_Y * ty + thread_y]);
        }
        // load B from shared memory to register
        #pragma unroll
        for (int thread_x = 0; thread_x < THREAD_SIZE_X; thread_x += 4) {
            FETCH_FLOAT4(frag_b[0][thread_x]) = FETCH_FLOAT4(Bs[load_stage_idx^1][0][THREAD_SIZE_X * tx + thread_x]);
        }
        //compute last tile mma THREAD_SIZE_X x THREAD_SIZE_Y
        #pragma unroll
        for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) {
            #pragma unroll
            for (int thread_x = 0; thread_x < THREAD_SIZE_X; ++thread_x) {
                accum[thread_y][thread_x] += frag_a[1][thread_y] * frag_b[1][thread_x];
            }
        }
    }while(tile_idx< K);

    // store back to C
    #pragma unroll
    for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) {
        #pragma unroll
        for (int thread_x = 0; thread_x < THREAD_SIZE_X; thread_x+=4) {
            FETCH_FLOAT4(C[OFFSET(
                BLOCK_SIZE_M * by + ty * THREAD_SIZE_Y + thread_y,
                BLOCK_SIZE_N * bx + tx * THREAD_SIZE_X + thread_x,
                N)]) = FETCH_FLOAT4(accum[thread_y][thread_x]);
        }
    }
}

// K: ldA
// N: ldB
template <
    const int BLOCK_SIZE_M,  // height of block of C that each thread block calculate
    const int BLOCK_SIZE_K,  // width of block of A that each thread block load into shared memory
    const int BLOCK_SIZE_N,  // width of block of C that each thread block calculate
    const int THREAD_SIZE_Y, // height of block of C that each thread calculate
    const int THREAD_SIZE_X,  // width of block of C that each thread calculate
    const bool ENABLE_DOUBLE_BUFFER // whether enable double buffering or not
    > 
__global__ void gemm_v2( 
    float * __restrict__ A,
    float * __restrict__ B,
    float * __restrict__ C, 
    const int M,
    const int N,
    const int K) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    // the threads number in Block of X,Y
    const int THREAD_X_PER_BLOCK = BLOCK_SIZE_N / THREAD_SIZE_X;
    const int THREAD_Y_PER_BLOCK = BLOCK_SIZE_M / THREAD_SIZE_Y;
    const int THREAD_NUM_PER_BLOCK = THREAD_X_PER_BLOCK * THREAD_Y_PER_BLOCK;

    // thread id in cur Block
    const int tid = ty * THREAD_X_PER_BLOCK + tx;

    // shared memory
    __shared__ float As[2][BLOCK_SIZE_K][BLOCK_SIZE_M];
    __shared__ float Bs[2][BLOCK_SIZE_K][BLOCK_SIZE_N];
    // registers for C
    float accum[THREAD_SIZE_Y][THREAD_SIZE_X];
    #pragma unroll
    for(int i=0; i<THREAD_SIZE_Y; i++){
        #pragma unroll
        for(int j=0; j<THREAD_SIZE_X; j++){
            accum[i][j]=0.0;
        }
    }
    // registers for A and B
    float frag_a[2][THREAD_SIZE_Y];
    float frag_b[2][THREAD_SIZE_X];
    // registers load global memory
    const int ldg_num_a = BLOCK_SIZE_M * BLOCK_SIZE_K / (THREAD_NUM_PER_BLOCK * 4);
    const int ldg_num_b = BLOCK_SIZE_K * BLOCK_SIZE_N / (THREAD_NUM_PER_BLOCK * 4);
    float ldg_a_reg[4*ldg_num_a];
    float ldg_b_reg[4*ldg_num_b];

    // threads number in one row
    const int A_TILE_THREAD_PER_ROW = BLOCK_SIZE_K / 4;
    const int B_TILE_THREAD_PER_ROW = BLOCK_SIZE_N / 4;

    // row number and col number that needs to be loaded by this thread
    const int A_TILE_ROW_START = tid / A_TILE_THREAD_PER_ROW;
    const int B_TILE_ROW_START = tid / B_TILE_THREAD_PER_ROW;

    const int A_TILE_COL = tid % A_TILE_THREAD_PER_ROW * 4; 
    const int B_TILE_COL = tid % B_TILE_THREAD_PER_ROW * 4;

    // row stride that thread uses to load multiple rows of a tile
    const int A_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / A_TILE_THREAD_PER_ROW;
    const int B_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / B_TILE_THREAD_PER_ROW;

    A = &A[(BLOCK_SIZE_M * by)* K];
    B = &B[BLOCK_SIZE_N * bx];

    //load index of the tile
    const int warp_id = tid / 32;
    const int lane_id = tid % 32;
    const int a_tile_index =  warp_id/2*16 + lane_id/8*4; //warp_id * 8 + (lane_id / 16)*4; // (warp_id/4)*32 + ((lane_id%16)/2)*4;
    const int b_tile_index =  warp_id%2*32 + lane_id%8*4; //(lane_id % 16) * 4; // (warp_id%4)*16 + (lane_id/16)*8 + (lane_id%2)*4;
    
    //transfer first tile from global mem to shared mem
    // load A from global memory to shared memory
    #pragma unroll
    for ( int i = 0 ; i < BLOCK_SIZE_M ; i += A_TILE_ROW_STRIDE) {
        int ldg_index = i / A_TILE_ROW_STRIDE * 4;
        FETCH_FLOAT4(ldg_a_reg[ldg_index]) = FETCH_FLOAT4(A[OFFSET(
            A_TILE_ROW_START + i, // row
            A_TILE_COL, // col
            K )]);
        As[0][A_TILE_COL][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index];
        As[0][A_TILE_COL+1][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+1];
        As[0][A_TILE_COL+2][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+2];
        As[0][A_TILE_COL+3][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+3];
    }
    // load B from global memory to shared memory
    #pragma unroll
    for ( int i = 0 ; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
        FETCH_FLOAT4(Bs[0][B_TILE_ROW_START + i][B_TILE_COL]) = FETCH_FLOAT4(B[OFFSET(
                B_TILE_ROW_START + i, // row
                B_TILE_COL, // col
                N )]);
    }
    __syncthreads();
    
    // load A from shared memory to register
    FETCH_FLOAT4(frag_a[0][0]) = FETCH_FLOAT4(As[0][0][a_tile_index]);
    FETCH_FLOAT4(frag_a[0][4]) = FETCH_FLOAT4(As[0][0][a_tile_index + 64]);
    
    // load B from shared memory to register
    FETCH_FLOAT4(frag_b[0][0]) = FETCH_FLOAT4(Bs[0][0][b_tile_index]);
    FETCH_FLOAT4(frag_b[0][4]) = FETCH_FLOAT4(Bs[0][0][b_tile_index + 64]);
    
    int write_stage_idx = 1;
    int tile_idx = 0;
    do{
        // next tile index
        tile_idx += BLOCK_SIZE_K;
        // load next tile from global mem
        if(tile_idx< K){
            #pragma unroll
            for ( int i = 0 ; i < BLOCK_SIZE_M ; i += A_TILE_ROW_STRIDE) {
                int ldg_index = i / A_TILE_ROW_STRIDE * 4;
                FETCH_FLOAT4(ldg_a_reg[ldg_index]) = FETCH_FLOAT4(A[OFFSET(
                    A_TILE_ROW_START + i, // row
                    A_TILE_COL + tile_idx, // col
                    K )]);
            }
            #pragma unroll
            for ( int i = 0 ; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
                int ldg_index = i / B_TILE_ROW_STRIDE * 4;
                FETCH_FLOAT4(ldg_b_reg[ldg_index]) = FETCH_FLOAT4(B[OFFSET(
                    tile_idx + B_TILE_ROW_START + i, // row
                    B_TILE_COL, // col
                    N )]);
            }
        }

        int load_stage_idx = write_stage_idx ^ 1;

        #pragma unroll
        for(int j=0; j<BLOCK_SIZE_K - 1; ++j){
            // load next tile from shared mem to register 
            // load A from shared memory to register
            FETCH_FLOAT4(frag_a[(j+1)%2][0]) = FETCH_FLOAT4(As[load_stage_idx][(j+1)][a_tile_index]);
            FETCH_FLOAT4(frag_a[(j+1)%2][4]) = FETCH_FLOAT4(As[load_stage_idx][(j+1)][a_tile_index + 64]);
            // load B from shared memory to register
            FETCH_FLOAT4(frag_b[(j+1)%2][0]) = FETCH_FLOAT4(Bs[load_stage_idx][(j+1)][b_tile_index]);
            FETCH_FLOAT4(frag_b[(j+1)%2][4]) = FETCH_FLOAT4(Bs[load_stage_idx][(j+1)][b_tile_index + 64]);
            // compute C THREAD_SIZE_X x THREAD_SIZE_Y
            #pragma unroll
            for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) {
                #pragma unroll
                for (int thread_x = 0; thread_x < THREAD_SIZE_X; ++thread_x) {
                    accum[thread_y][thread_x] += frag_a[j%2][thread_y] * frag_b[j%2][thread_x];
                }
            }
        }

        if(tile_idx < K){
            // load A from global memory to shared memory
            #pragma unroll
            for ( int i = 0 ; i < BLOCK_SIZE_M ; i += A_TILE_ROW_STRIDE) {
                int ldg_index = i / A_TILE_ROW_STRIDE * 4;
                As[write_stage_idx][A_TILE_COL][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index];
                As[write_stage_idx][A_TILE_COL+1][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+1];
                As[write_stage_idx][A_TILE_COL+2][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+2];
                As[write_stage_idx][A_TILE_COL+3][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+3];
            }
            // load B from global memory to shared memory
            #pragma unroll
            for ( int i = 0 ; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
                int ldg_index = i / B_TILE_ROW_STRIDE * 4;
                FETCH_FLOAT4(Bs[write_stage_idx][B_TILE_ROW_START + i][B_TILE_COL]) = FETCH_FLOAT4(ldg_b_reg[ldg_index]);
            }
            // use double buffer, only need one sync
            __syncthreads();
            // switch
            write_stage_idx ^= 1;
        }

        // load first tile from shared mem to register of next iter
        // load A from shared memory to register
        FETCH_FLOAT4(frag_a[0][0]) = FETCH_FLOAT4(As[load_stage_idx^1][0][a_tile_index]);
        FETCH_FLOAT4(frag_a[0][4]) = FETCH_FLOAT4(As[load_stage_idx^1][0][a_tile_index + 64]);
        // load B from shared memory to register
        FETCH_FLOAT4(frag_b[0][0]) = FETCH_FLOAT4(Bs[load_stage_idx^1][0][b_tile_index]);
        FETCH_FLOAT4(frag_b[0][4]) = FETCH_FLOAT4(Bs[load_stage_idx^1][0][b_tile_index + 64]);
        // compute C THREAD_SIZE_X x THREAD_SIZE_Y
        #pragma unroll
        for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) {
            #pragma unroll
            for (int thread_x = 0; thread_x < THREAD_SIZE_X; ++thread_x) {
                accum[thread_y][thread_x] += frag_a[1][thread_y] * frag_b[1][thread_x];
            }
        }
    }while(tile_idx< K);
    
    const int c_block_row = a_tile_index;
    const int c_block_col = b_tile_index;

    //store C00 block
    for(int i=0; i<4; i++){
      FETCH_FLOAT4(C[OFFSET(
        BLOCK_SIZE_M * by + c_block_row + i,
        BLOCK_SIZE_N * bx + c_block_col,
        N)]) = FETCH_FLOAT4(accum[i][0]);
    }
    //store C01 block
    for(int i=0; i<4; i++){
      FETCH_FLOAT4(C[OFFSET(
        BLOCK_SIZE_M * by + c_block_row + i,
        BLOCK_SIZE_N * bx + c_block_col + 64,
        N)]) = FETCH_FLOAT4(accum[i][4]);
    }
    //store C10 block
    for(int i=0; i<4; i++){
      FETCH_FLOAT4(C[OFFSET(
        BLOCK_SIZE_M * by + c_block_row + 64 + i,
        BLOCK_SIZE_N * bx + c_block_col,
        N)]) = FETCH_FLOAT4(accum[i+4][0]);
    }
    //store C11 block
    for(int i=0; i<4; i++){
      FETCH_FLOAT4(C[OFFSET(
        BLOCK_SIZE_M * by + c_block_row + 64 + i,
        BLOCK_SIZE_N * bx + c_block_col + 64,
        N)]) = FETCH_FLOAT4(accum[i+4][4]);
    }
}


int main(int argc, char** argv) {
    if (argc != 4) {
        printf("usage: ./main [M] [K] [N]\n");
        exit(0);
    }
    size_t M = atoi(argv[1]);
    size_t K = atoi(argv[2]);
    size_t N = atoi(argv[3]);

    assert( M%8 == 0); 
    assert( N%8 == 0); 
    assert( K%8 == 0); 

    size_t bytes_A = sizeof(float) * M * K;
    size_t bytes_B = sizeof(float) * K * N;
    size_t bytes_C = sizeof(float) * M * N;
    float* h_A = (float*)malloc(bytes_A);
    float* h_B = (float*)malloc(bytes_B);
    float* h_C_blas = (float*)malloc(bytes_C);

    float* d_A;
    float* d_B;
    float* d_C;

    checkCudaErrors(hipMalloc(&d_A, bytes_A));
    checkCudaErrors(hipMalloc(&d_B, bytes_B));
    checkCudaErrors(hipMalloc(&d_C, bytes_C));
    
    double flopsPerMatrixMul = 2.0 * M * N * K;

    const int BLOCK_SIZE_M = 128;
    const int BLOCK_SIZE_K = 8;
    const int BLOCK_SIZE_N = 128;
    const int THREAD_SIZE_X = 8;
    const int THREAD_SIZE_Y = 8;
    const bool ENABLE_DOUBLE_BUFFER = false;

    // generate A
    for( int i = 0; i < M * K; i++ ){
        h_A[i] = i / 13;
    }

    // generate B
    for( int i = 0; i < K * N; i++ ) {
        h_B[i] = i % 13;
    }

    checkCudaErrors(hipMemcpy( d_A, h_A, bytes_A, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy( d_B, h_B, bytes_B, hipMemcpyHostToDevice));
    
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    float msecTotal = 0;
    int nIter = 1000;

    // cublas
    double msecPerMatrixMul_cublas = 0;
    double gigaFlops_cublas = 0;
    hipblasHandle_t blas_handle;  
    hipblasCreate(&blas_handle);
    float alpha = 1.0;
    float beta = 0;
    checkCudaErrors(hipMemcpy( d_C, h_C_blas, bytes_C, hipMemcpyHostToDevice));
    checkCudaErrors(hipEventRecord(start));
    for (int run = 0 ; run < nIter; run ++ ) {
        hipblasSgemm (blas_handle, HIPBLAS_OP_T, HIPBLAS_OP_T, 
            M, N, K, &alpha, 
            d_A, K, d_B, N, &beta, d_C, N
        );
    }
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    checkCudaErrors(hipMemcpy( h_C_blas, d_C, bytes_C, hipMemcpyDeviceToHost));

    msecPerMatrixMul_cublas = msecTotal / nIter;
    gigaFlops_cublas = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul_cublas / 1000.0f);
    printf( "CuBlas Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
        gigaFlops_cublas,
        msecPerMatrixMul_cublas,
        flopsPerMatrixMul);

    hipblasDestroy(blas_handle); 

    //base version
    double msecPerMatrixMul_v0 = 0;
    double gigaFlops_v0 = 0;

    float* h_C0 = (float*)malloc(bytes_C);
    memset(h_C0, 0, bytes_C);
    checkCudaErrors(hipMemcpy( d_C, h_C0, bytes_C, hipMemcpyHostToDevice));
    checkCudaErrors(hipEventRecord(start));
    for (int run = 0 ; run < nIter; run ++ ) {
        dim3 dimBlock(32, 32);
        dim3 dimGrid(N / dimBlock.x, M / dimBlock.y);
        gemm_base<<< dimGrid, dimBlock >>>(d_A, d_B, d_C, M, N, K);
    }
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));
    checkCudaErrors(hipMemcpy( h_C0, d_C, bytes_C, hipMemcpyDeviceToHost));

    msecPerMatrixMul_v0 = msecTotal / nIter;
    gigaFlops_v0 = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul_v0 / 1000.0f);
    printf( "gemm base Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
        gigaFlops_v0,
        msecPerMatrixMul_v0,
        flopsPerMatrixMul);

    double eps = 1.e-6;  // machine zero
    bool correct = true;
    for (int i = 0; i < M * N; i++) {
        int row = i / N;
        int col = i % N;
        double abs_err = fabs(h_C0[i] - h_C_blas[col * M + row]);
        double dot_length = M;
        double abs_val = fabs(h_C0[i]);
        double rel_err = abs_err / abs_val / dot_length;
        if (rel_err > eps) {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
                    i, h_C0[i], h_C_blas[col * M + row], eps);
            correct = false;
            break;
        }
    }

    printf("base version %s\n", correct ? "Result= PASS" : "Result= FAIL");
    printf("ratio= %f\n", gigaFlops_v0 / gigaFlops_cublas);

    //version 1
    double msecPerMatrixMul_v1 = 0;
    double gigaFlops_v1 = 0;
    float* h_C1 = (float*)malloc(bytes_C);
    memset(h_C1, 0, bytes_C);
    checkCudaErrors(hipMemcpy( d_C, h_C1, bytes_C, hipMemcpyHostToDevice));
    checkCudaErrors(hipEventRecord(start));
    for (int run = 0 ; run < nIter; run ++ ) {
        dim3 dimBlock(BLOCK_SIZE_N / THREAD_SIZE_X, BLOCK_SIZE_M / THREAD_SIZE_Y);
        dim3 dimGrid(N / BLOCK_SIZE_N, M / BLOCK_SIZE_M);
        gemm_v1<BLOCK_SIZE_M, BLOCK_SIZE_K, BLOCK_SIZE_N, THREAD_SIZE_Y, THREAD_SIZE_X, ENABLE_DOUBLE_BUFFER> 
        <<< dimGrid, dimBlock >>>(d_A, d_B, d_C, M, N, K);
    }
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));


    checkCudaErrors(hipMemcpy( h_C1, d_C, bytes_C, hipMemcpyDeviceToHost));

    msecPerMatrixMul_v1 = msecTotal / nIter;
    gigaFlops_v1 = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul_v1 / 1000.0f);
    printf( "gemm v1 Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
        gigaFlops_v1,
        msecPerMatrixMul_v1,
        flopsPerMatrixMul);

    eps = 1.e-6;  // machine zero
    correct = true;
    for (int i = 0; i < M * N; i++) {
        int row = i / N;
        int col = i % N;
        double abs_err = fabs(h_C1[i] - h_C_blas[col * M + row]);
        double dot_length = M;
        double abs_val = fabs(h_C1[i]);
        double rel_err = abs_err / abs_val / dot_length;
        if (rel_err > eps) {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
                    i, h_C1[i], h_C_blas[col * M + row], eps);
            correct = false;
            break;
        }
    }

    printf("version 1 %s\n", correct ? "Result= PASS" : "Result= FAIL");
    printf("ratio= %f\n", gigaFlops_v1 / gigaFlops_cublas);

    //version 2
    double msecPerMatrixMul_v2 = 0;
    double gigaFlops_v2 = 0;
    float* h_C2 = (float*)malloc(bytes_C);
    memset(h_C2, 0, bytes_C);
    checkCudaErrors(hipMemcpy( d_C, h_C2, bytes_C, hipMemcpyHostToDevice));
    checkCudaErrors(hipEventRecord(start));
    for (int run = 0 ; run < nIter; run ++ ) {
        dim3 dimBlock(BLOCK_SIZE_N / THREAD_SIZE_X, BLOCK_SIZE_M / THREAD_SIZE_Y);
        dim3 dimGrid(N / BLOCK_SIZE_N, M / BLOCK_SIZE_M);
        gemm_v2<BLOCK_SIZE_M, BLOCK_SIZE_K, BLOCK_SIZE_N, THREAD_SIZE_Y, THREAD_SIZE_X, ENABLE_DOUBLE_BUFFER> 
        <<< dimGrid, dimBlock >>>(d_A, d_B, d_C, M, N, K);
    }
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));


    checkCudaErrors(hipMemcpy( h_C2, d_C, bytes_C, hipMemcpyDeviceToHost));

    msecPerMatrixMul_v2 = msecTotal / nIter;
    gigaFlops_v2 = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul_v2 / 1000.0f);
    printf( "gemm v2 Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
        gigaFlops_v2,
        msecPerMatrixMul_v2,
        flopsPerMatrixMul);

    eps = 1.e-6;  // machine zero
    correct = true;
    for (int i = 0; i < M * N; i++) {
        int row = i / N;
        int col = i % N;
        double abs_err = fabs(h_C1[i] - h_C_blas[col * M + row]);
        double dot_length = M;
        double abs_val = fabs(h_C1[i]);
        double rel_err = abs_err / abs_val / dot_length;
        if (rel_err > eps) {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
                    i, h_C1[i], h_C_blas[col * M + row], eps);
            correct = false;
            break;
        }
    }

    printf("version 2 %s\n", correct ? "Result= PASS" : "Result= FAIL");
    printf("ratio= %f\n", gigaFlops_v2 / gigaFlops_cublas);
    
    hipEventDestroy(stop);
    hipEventDestroy(start);

    // Free Memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    free(h_A);
    free(h_B);
    free(h_C_blas);
    free(h_C0);
    free(h_C1);
    free(h_C2);
}
